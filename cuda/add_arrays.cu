#include "hip/hip_runtime.h"
// https://devblogs.nvidia.com/even-easier-introduction-cuda
// https://devblogs.nvidia.com/unified-memory-cuda-beginners

#include <iostream>
#include <math.h>


// cuda kernel
__global__
void add(size_t num_elements, const float* x, float* result)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;
  for (size_t i = index; i < num_elements; i += stride)
  {
    result[i] += x[i];
  }
}


int main()
{
  const size_t num_elements = 1 << 20;  // 1M elements
  float* x;
  float* y;
  hipMallocManaged(&x, num_elements * sizeof(float));
  hipMallocManaged(&y, num_elements * sizeof(float));

  for (size_t i = 0; i < num_elements; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // prefetch data to GPU
  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(x, num_elements * sizeof(float), device, NULL);
  hipMemPrefetchAsync(y, num_elements * sizeof(float), device, NULL);

  size_t block_size = 256;
  size_t num_blocks = (num_elements + block_size - 1) / block_size;
  // run kernel on GPU
  add<<<num_blocks, block_size>>>(num_elements, x, y);
  // wait for GPU to finish
  hipDeviceSynchronize();

  float max_error = 0.0f;
  for (size_t i = 0; i < num_elements; i++)
  {
    max_error = fmax(max_error, fabs(y[i] - 3.0f));
  }
  std::cout << "max error: " << max_error << std::endl;

  hipFree(x);
  hipFree(y);
}
